#include "hip/hip_runtime.h"
#include <opencv2/features2d.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>
#include <string> 
#include <iomanip>
//#include "opencv2/line_descriptor/descriptor.hpp"
#include <fstream>
#include <sqlite3.h>
#include <vector>
#include <unistd.h>
#include <algorithm>
#include <Eigen/Dense>
#include <iterator>

#define CALL_SQLITE(f)                                          \
    {                                                           \
        int i;                                                  \
        i = sqlite3_ ## f;                                      \
        if (i != SQLITE_OK) {                                   \
            fprintf (stderr, "%s failed with status %d: %s\n",  \
                     #f, i, sqlite3_errmsg (db));               \
            exit (1);                                           \
        }                                                       \
    }                                                           \

#define CALL_SQLITE_EXPECT(f,x)                                 \
    {                                                           \
        int i;                                                  \
        i = sqlite3_ ## f;                                      \
        if (i != SQLITE_ ## x) {                                \
            fprintf (stderr, "%s failed with status %d: %s\n",  \
                     #f, i, sqlite3_errmsg (db));               \
            exit (1);                                           \
        }                                                       \
    }                                                           \


#define MINIMUM_THRESH 1.25

using namespace cv;
using namespace std;

vector<Eigen::MatrixXf> database;

static int callback(void *NotUsed, int argc, char **argv, char **azColName) {
	float tempFloat;
	const char *buffer = argv[3];

	int numRows = atoi(argv[1]);

	int image_id = atoi(argv[0]);
	Eigen::MatrixXf descriptors = Eigen::MatrixXf::Zero(numRows, 128);

	for (int i = 0; i < numRows; i++) 
	{
		for (int j = 0; j < 128; j++) {
		  // cout << i << " " << j << endl;
		  memcpy(&tempFloat, buffer + 4 * j + (4 * i * 128), sizeof(float));
		  descriptors(i, j) = tempFloat;
		}
	}

	cout << "Loading Image: " << image_id << endl;

	database.push_back(descriptors.transpose()); // .data() is returned as column major

	return 0;
}

sqlite3_int64 getPairID(sqlite3_int64 id1, sqlite3_int64 id2) {
	sqlite3_int64 pairID;
  if (id1 > id2) {
    pairID = 2147483647 * id2 + id1;
  } else {
    pairID = 2147483647 * id1 + id2;
  }

  return pairID;
}

void getImageIDs(long pairID, int &id1, int &id2) {
  id2 = pairID % 2147483647;
  id1 = (pairID - id2) / 2147483647;
}

__global__  void cuFindMatches(float* im1, float* im2, int* matches, float* SSDs, int rows1, int rows2) {
	float ssd;
	float diff;
	__shared__ float best;
	__shared__ int bestIndx;


    if (threadIdx.x == 0) {

      best = MINIMUM_THRESH;

      bestIndx = -1;


    }

    __syncthreads();


	int idx = threadIdx.x;
	int stride = blockDim.x;

	// Get initial matches
	//for(int i =Xindex; i<rows1; i += Xstride){
		for(int j = idx; j<rows2; j +=stride){
			ssd = 0;
			for(int k = 0; k<128; k++){
				diff = im1[k + 128*blockIdx.x] - im2[k + 128*j];
				ssd += diff*diff;
			}

			ssd = sqrt(ssd);

			//__syncthreads();

			if(ssd < best){
				best = ssd;
				bestIndx = j;
			}
		}
	//}


	matches[blockIdx.x] = bestIndx;
	SSDs[blockIdx.x] = best;

	__syncthreads();

	if(matches[blockIdx.x] >= 0){

		// // Verify best matches in other image
		// for(int ii=idx; ii<rows2; ii +=stride){
		// 	//for(int jj=Xindex; jj<rows1; jj += Xstride){
		// 		ssd = 0;
		// 		for(int kk=0; kk<128; kk++){
		// 			diff = im2[kk + 128*ii] - im1[kk + 128*blockIdx.x];
		// 			ssd += diff*diff;
		// 		}
		// 		ssd = sqrt(ssd);

		// 		if((ssd <= SSDs[blockIdx.x]) && (ii != matches[blockIdx.x])){
		// 			matches[blockIdx.x] = -1;
		// 		}
		// 	//}
		// }

		__syncthreads();
		if(matches[blockIdx.x] >= 0){

			// Remove duplicates
			//for(int iii = Yindex; iii<rows1; iii += Ystride){
				for(int jjj = idx; jjj<rows1; jjj += stride){
					if(blockIdx.x != jjj){
						if(matches[blockIdx.x] == matches[jjj]){
							matches[blockIdx.x] = -1;
							matches[jjj] = -1;
						}
					}
				}
			//}

		}

	}


}


int main( int argc, char** argv )
{
	char *zErrMsg = 0;
	//const char *dbFilename = "style_cup_194.db";
	const char *dbFilename = "glove_mosaic_learned.db";
	sqlite3 *db;
	char * sql;

	CALL_SQLITE (open (dbFilename, &db));

	sql = "SELECT * from learned";
	CALL_SQLITE (exec(db, sql, callback, 0, &zErrMsg));


	float *d_im1, *d_im2, *d_SSDs;
	int *d_matches;
    float *im1;
    float *im2;
    sqlite3_stmt * stmt = NULL;

	float *SSDs1;
	int *matches1;
	int *output_buffer;

	int rows1, rows2;

    vector<int> blobData;
    sqlite3_int64 pairID;

    int config = 0;

    for(int i=108; i<185;i++)
    {
    	im1 = database[i].data();
    	rows1 = database[i].cols();
    	hipMalloc((void **)&d_im1,rows1*128*4);
    	hipMemcpy(d_im1, im1, rows1*128*4, hipMemcpyHostToDevice);

		hipMalloc((void **)&d_matches,rows1*4);
		hipMalloc((void **)&d_SSDs,rows1*4);

	    SSDs1 = new float[rows1];
	    matches1 = new int[rows1];
  
		for(int j = i; j<185; j++)
		{
				if((i!=j) && (j < i + 30)){

			    	im2 = database[j].data();
			    	rows2 = database[j].cols();
			    	hipMalloc((void **)&d_im2,rows2*128*4);
		    		hipMemcpy(d_im2, im2, rows2*128*4, hipMemcpyHostToDevice);

					cout << "Processing Image " << i + 1 << " and Image " << j + 1 << endl;
					//cout << "rows1 " << rows1 << " rows2 " << rows2 << endl;

					hipMemset(d_SSDs, MINIMUM_THRESH, rows1*4);

				    cuFindMatches<<<rows1, 1024>>>(d_im1,d_im2,d_matches,d_SSDs,rows1,rows2);

				    hipMemcpy(matches1, d_matches, rows1*sizeof(int), hipMemcpyDeviceToHost);
				    hipMemcpy(SSDs1, d_SSDs, rows1*sizeof(int), hipMemcpyDeviceToHost);


				    for(int k = 0; k<rows1; k++)
				    {
				    	//printf("%d matches1 %d ssd1 %f matches2 %d ssd2 %f\n", k, matches1[k], SSDs1[k], matches2[k], SSDs2[k]);
				    	if(matches1[k]>=0)//&&(matches2[matches1[k]] == k))
				    	{

							blobData.push_back(k);
							blobData.push_back(matches1[k]);
					    }
				    }


				    cout << "[VERIFIED] " << blobData.size()/2 << " Matches " << endl;

				    output_buffer = new int[blobData.size()];

				    std::copy(blobData.begin(), blobData.end(), output_buffer);

				    // int *pt = &blobData[0][0];
				    
				    // for(int l=0;l<blobData.size();l++){
			    	// 	// printf("bufffer i: %d, j: %d\n",*(pt + l*2*4), *(pt + 4 + l*2*4));
			    	// 	//printf("vector i: %d, j: %d\n",blobData[l][0], blobData[l][1]);
			    	// 	output_buffer[l][0] = blobData[l][0];
			    	// 	output_buffer[l][1] = blobData[l][1];
				    // }

				    //cout << blobData.size()*2*4 << endl;
				    config = 0;
				    if(blobData.size()/2 > 0){
				    	if(blobData.size()/2 > 100){
				    		config = 6;
				    	}else{
				    		config = 3;
				    	}
				    }

				    //cout << "pair id " << pairID << endl;
				    pairID = getPairID((sqlite3_int64)(i+1), (sqlite3_int64)(j+1));

				    CALL_SQLITE (prepare_v2 (db, "INSERT INTO matches (pair_id, rows, cols, data) VALUES (?, ?, '2', ? )", -1, &stmt, NULL));
				    CALL_SQLITE (bind_int64(stmt, 1, pairID));
				    CALL_SQLITE (bind_int(stmt, 2, blobData.size()/2));
				    CALL_SQLITE (bind_blob(stmt, 3, output_buffer,blobData.size()*4,SQLITE_STATIC));

				    CALL_SQLITE_EXPECT (step (stmt), DONE);
				    printf ("Wrote data to row id %lld\n", sqlite3_last_insert_rowid (db));

				    CALL_SQLITE (prepare_v2 (db, "INSERT INTO inlier_matches (pair_id, rows, cols, data, config) VALUES (?, ?, '2', ?, ? )", -1, &stmt, NULL));
				    CALL_SQLITE (bind_int64(stmt, 1, pairID));
				    CALL_SQLITE (bind_int(stmt, 2, blobData.size()/2));
				    CALL_SQLITE (bind_blob(stmt, 3, output_buffer,blobData.size()*4,SQLITE_STATIC));
				    CALL_SQLITE (bind_int(stmt, 4, config));

				    CALL_SQLITE_EXPECT (step (stmt), DONE);
				    printf ("Wrote data to row id %lld\n", sqlite3_last_insert_rowid (db));

				    sqlite3_finalize(stmt);

				    blobData.clear();

					hipFree(d_im2);

					delete[] output_buffer;
			}

		}

		delete[] SSDs1;
		delete[] matches1;

		hipFree(d_im1);
		hipFree(d_matches);
		hipFree(d_SSDs);
	}

	hipFree(d_im1);
	hipFree(d_im2);
	hipFree(d_matches);
	hipFree(d_SSDs);


	CALL_SQLITE (close(db));
}
