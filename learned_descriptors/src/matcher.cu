#include "hip/hip_runtime.h"
#include <opencv2/features2d.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>
#include <string> 
#include <iomanip>
//#include "opencv2/line_descriptor/descriptor.hpp"
#include <fstream>
#include <sqlite3.h>
#include <vector>
#include <unistd.h>
#include <algorithm>
#include <Eigen/Dense>
#include <iterator>

#define CALL_SQLITE(f)                                          \
    {                                                           \
        int i;                                                  \
        i = sqlite3_ ## f;                                      \
        if (i != SQLITE_OK) {                                   \
            fprintf (stderr, "%s failed with status %d: %s\n",  \
                     #f, i, sqlite3_errmsg (db));               \
            exit (1);                                           \
        }                                                       \
    }                                                           \

#define CALL_SQLITE_EXPECT(f,x)                                 \
    {                                                           \
        int i;                                                  \
        i = sqlite3_ ## f;                                      \
        if (i != SQLITE_ ## x) {                                \
            fprintf (stderr, "%s failed with status %d: %s\n",  \
                     #f, i, sqlite3_errmsg (db));               \
            exit (1);                                           \
        }                                                       \
    }                                                           \


#define MINIMUM_THRESH 1.25

using namespace cv;
using namespace std;

vector<Eigen::MatrixXf> database;
int image_count;

// Callback for loading descriptors from database
static int callback(void *NotUsed, int argc, char **argv, char **azColName) {
	float tempFloat;
	const char *buffer = argv[3];

	int numRows = atoi(argv[1]);

	int image_id = atoi(argv[0]);
	Eigen::MatrixXf descriptors = Eigen::MatrixXf::Zero(numRows, 128);

	for (int i = 0; i < numRows; i++) 
	{
		for (int j = 0; j < 128; j++) {
		  memcpy(&tempFloat, buffer + 4 * j + (4 * i * 128), sizeof(float));
		  descriptors(i, j) = tempFloat;
		}
	}

	cout << "Loading Image: " << image_id << endl;
	image_count++;
	database.push_back(descriptors.transpose()); // .data() is returned as column major

	return 0;
}

sqlite3_int64 getPairID(sqlite3_int64 id1, sqlite3_int64 id2) {
	sqlite3_int64 pairID;
  if (id1 > id2) {
    pairID = 2147483647 * id2 + id1;
  } else {
    pairID = 2147483647 * id1 + id2;
  }

  return pairID;
}

void getImageIDs(long pairID, int &id1, int &id2) {
  id2 = pairID % 2147483647;
  id1 = (pairID - id2) / 2147483647;
}

__global__  void cuFindMatches(float* im1, float* im2, int* matches, float* SSDs, int rows1, int rows2) {
	float ssd;
	float diff;
	__shared__ float best;
	__shared__ int bestIndx;

    if (threadIdx.x == 0) {

      best = MINIMUM_THRESH;
      bestIndx = -1;
    }

    __syncthreads();

	int idx = threadIdx.x;
	int stride = blockDim.x;

	// Get initial matches
	for(int ii = idx; ii<rows2; ii +=stride){
		ssd = 0;
		for(int k = 0; k<128; k++){
			diff = im1[k + 128*blockIdx.x] - im2[k + 128*ii];
			ssd += diff*diff;
		}

		ssd = sqrt(ssd);

		if(ssd < best){
			best = ssd;
			bestIndx = ii;
		}
	}


	matches[blockIdx.x] = bestIndx;
	SSDs[blockIdx.x] = best;

	__syncthreads();
	if(matches[blockIdx.x] >= 0){
		__syncthreads();
		if(matches[blockIdx.x] >= 0){
			// Remove duplicates and non mutual matches
			for(int jj = idx; jj<rows1; jj += stride){
				if(blockIdx.x != jj){
					if(matches[blockIdx.x] == matches[jj]){
						matches[blockIdx.x] = -1;
						matches[jj] = -1;
					}
				}
			}
		}
	}


}


int main( int argc, char** argv )
{
	char *zErrMsg = 0;
	//const char *dbFilename = "style_cup_194.db";
	const char *dbFilename = "glove_mosaic_learned.db";
	sqlite3 *db;
	char * sql;

	image_count = 0;

	// Read descriptors from database
	CALL_SQLITE (open (dbFilename, &db));
	sql = "SELECT * from learned";
	CALL_SQLITE (exec(db, sql, callback, 0, &zErrMsg));


	float *d_im1, *d_im2, *d_SSDs;
	int *d_matches;
    float *im1;
    float *im2;
    sqlite3_stmt * stmt = NULL;
	float *SSDs1;
	int *matches1;
	int *output_buffer;
	int rows1, rows2;

    vector<int> blobData;
    sqlite3_int64 pairID;

    int config = 0;

    for(int i=0; i<image_count;i++)
    {
    	im1 = database[i].data();
    	rows1 = database[i].cols();
    	hipMalloc((void **)&d_im1,rows1*128*4);
    	hipMemcpy(d_im1, im1, rows1*128*4, hipMemcpyHostToDevice);

		hipMalloc((void **)&d_matches,rows1*4);
		hipMalloc((void **)&d_SSDs,rows1*4);

	    SSDs1 = new float[rows1];
	    matches1 = new int[rows1];
  
		for(int j = i; j<image_count; j++)
		{
			if((i!=j) && (j < i + 30)){ // Only comparing images within 30 frames

		    	im2 = database[j].data();
		    	rows2 = database[j].cols();
		    	hipMalloc((void **)&d_im2,rows2*128*4);
	    		hipMemcpy(d_im2, im2, rows2*128*4, hipMemcpyHostToDevice);

				cout << "Processing Image " << i + 1 << " and Image " << j + 1 << endl;

				hipMemset(d_SSDs, MINIMUM_THRESH, rows1*4);

			    cuFindMatches<<<rows1, 1024>>>(d_im1,d_im2,d_matches,d_SSDs,rows1,rows2);

			    hipMemcpy(matches1, d_matches, rows1*sizeof(int), hipMemcpyDeviceToHost);
			    hipMemcpy(SSDs1, d_SSDs, rows1*sizeof(int), hipMemcpyDeviceToHost);


			    for(int k = 0; k<rows1; k++)
			    {
			    	//printf("%d matches1 %d ssd1 %f matches2 %d ssd2 %f\n", k, matches1[k], SSDs1[k], matches2[k], SSDs2[k]);
			    	if(matches1[k]>=0)
			    	{
						blobData.push_back(k);
						blobData.push_back(matches1[k]);
				    }
			    }

			    cout << "[VERIFIED] " << blobData.size()/2 << " Matches " << endl;

			    output_buffer = new int[blobData.size()];

			    std::copy(blobData.begin(), blobData.end(), output_buffer);

			    //cout << blobData.size()*2*4 << endl;
			    config = 0;
			    if(blobData.size()/2 > 0){
			    	if(blobData.size()/2 > 100){
			    		config = 6;
			    	}else{
			    		config = 3;
			    	}
			    }

			    // Save matches to database, COLMAP only actually looks at inlier_matches
			    pairID = getPairID((sqlite3_int64)(i+1), (sqlite3_int64)(j+1));

			    CALL_SQLITE (prepare_v2 (db, "INSERT INTO matches (pair_id, rows, cols, data) VALUES (?, ?, '2', ? )", -1, &stmt, NULL));
			    CALL_SQLITE (bind_int64(stmt, 1, pairID));
			    CALL_SQLITE (bind_int(stmt, 2, blobData.size()/2));
			    CALL_SQLITE (bind_blob(stmt, 3, output_buffer,blobData.size()*4,SQLITE_STATIC));

			    CALL_SQLITE_EXPECT (step (stmt), DONE);
			    printf ("Wrote data to row id %lld\n", sqlite3_last_insert_rowid (db));

			    CALL_SQLITE (prepare_v2 (db, "INSERT INTO inlier_matches (pair_id, rows, cols, data, config) VALUES (?, ?, '2', ?, ? )", -1, &stmt, NULL));
			    CALL_SQLITE (bind_int64(stmt, 1, pairID));
			    CALL_SQLITE (bind_int(stmt, 2, blobData.size()/2));
			    CALL_SQLITE (bind_blob(stmt, 3, output_buffer,blobData.size()*4,SQLITE_STATIC));
			    CALL_SQLITE (bind_int(stmt, 4, config));

			    CALL_SQLITE_EXPECT (step (stmt), DONE);
			    printf ("Wrote data to row id %lld\n", sqlite3_last_insert_rowid (db));

			    sqlite3_finalize(stmt);

			    blobData.clear();

				hipFree(d_im2);

				delete[] output_buffer;
			}

		}

		delete[] SSDs1;
		delete[] matches1;

		hipFree(d_im1);
		hipFree(d_matches);
		hipFree(d_SSDs);
	}

	hipFree(d_im1);
	hipFree(d_im2);
	hipFree(d_matches);
	hipFree(d_SSDs);


	CALL_SQLITE (close(db));
}
